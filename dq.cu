#include <iostream>
#include <thrust/system_error.h>
#include <thrust/version.h>
#if THRUST_VERSION < 100600
#include <thrust/system/cuda_error.h>
#else
#include <thrust/system/cuda/error.h>
#endif

void query_all(std::ostream &os)
{
  int device_count = 0;

  hipError_t error = hipGetDeviceCount(&device_count);
  if(error) throw thrust::system_error(error, thrust::cuda_category());

  switch(device_count)
  {
    case 0:
    {
      os << "There is no device supporting CUDA." << std::endl;
      break;
    }

    case 1:
    {
      os << "There is 1 device supporting CUDA." << std::endl;
      break;
    }

    default:
    {
      os << "There are " << device_count << " devices supporting CUDA." << std::endl;
      break;
    }
  }

  for(int dev = 0; dev < device_count; ++dev)
  {
    hipDeviceProp_t device_prop;
    hipError_t error = hipGetDeviceProperties(&device_prop, dev);
    if(error) throw thrust::system_error(error, thrust::cuda_category());
    
    os << std::endl;
    os << "Device " << dev << ": \"" << device_prop.name << "\"" << std::endl;
    os << "  Major revision number:                         " << device_prop.major << std::endl;
    os << "  Minor revision number:                         " << device_prop.minor << std::endl;
    os << "  Total amount of global memory:                 " << device_prop.totalGlobalMem << " bytes" << std::endl;
    os << "  Number of multiprocessors:                     " << device_prop.multiProcessorCount << std::endl;
    os << "  Total amount of constant memory:               " << device_prop.totalConstMem << " bytes" << std::endl;
    os << "  Total amount of shared memory per block:       " << device_prop.sharedMemPerBlock << " bytes" << std::endl;
    os << "  Total number of registers available per block: " << device_prop.regsPerBlock << std::endl;
    os << "  Warp size:                                     " << device_prop.warpSize << std::endl;
    os << "  Maximum number of threads per block:           " << device_prop.maxThreadsPerBlock << std::endl;
    os << "  Maximum sizes of each dimension of a block:    " << device_prop.maxThreadsDim[0] << " " << device_prop.maxThreadsDim[1] << " " << device_prop.maxThreadsDim[2] << std::endl;
    os << "  Maximum sizes of each dimension of a grid:     " << device_prop.maxGridSize[0] << " " << device_prop.maxGridSize[1] << " " << device_prop.maxGridSize[2] << std::endl;
    os << "  Maximum memory pitch:                          " << device_prop.memPitch << std::endl;
    os << "  Texture alignment:                             " << device_prop.textureAlignment << " bytes" << std::endl;
    os << "  Clock rate:                                    " << device_prop.clockRate * 1e-6f << " GHz" << std::endl;
    os << "  Concurrent copy and execution:                 " << (device_prop.deviceOverlap ? "Yes" : "No") << std::endl;
  }
}

int main()
{
  try
  {
    query_all(std::cout);
  }
  catch(std::runtime_error &e)
  {
    std::cerr << "main(): caught exception: " << e.what() << std::endl;
  }

  return 0;
}

